#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <ctime>
#include <string>
#include <cmath>
#include <omp.h>
#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "error.cuh"
#include <fstream>
#include <sstream>
#include <unordered_map>
#include <vector>
#include <chrono>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>

#define _USE_MATH_DEFINES
#define EXP 0.0000000000

using namespace std;
using Complex = thrust::complex<float>;

// complexExp 函数的实现
__device__ thrust::complex<float> complexExp(const Complex &d) {
    float realPart = exp(d.real()) * cos(d.imag());
    float imagPart = exp(d.real()) * sin(d.imag());
    return thrust::complex<float>(realPart, imagPart);
}
// complexAbs 函数的实现
__device__ thrust::complex<float> ComplexAbs(const Complex &d) {
    // 复数的模定义为 sqrt(real^2 + imag^2)
    return thrust::complex<float>(sqrt(d.real() * d.real() + d.imag() * d.imag()));
}

struct timeval start, finish;
float total_time;

string address = "./frequency_1M/";
string F_address = "./F_recon_1M/";
string para;
string duration = "frequency1M";  // 第几个周期的uvw
string sufix = ".txt";

// 1 M
const int uvw_presize = 4000000;


// 定义计算C的核函数，每一个线程处理一个q的值，q为0-nn的范围， 但是NX中保存的索引是1-nn，因此需要对齐，验证正确
__global__ void computeC(float *NX, float *FF, float *C, float *dN, int nn, int NX_size) {
    int q = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (q <= nn){
        float sumReal = 0.0;
        int count=0;
        for (int i=0; i < NX_size; ++i) {
            if (NX[i] == q) {
                sumReal += FF[i];
                count++;
            }
        }
        if (count > 0) {
            C[q-1] = sumReal/count;
        }
    }
}

// 定义计算可见度核函数, 验证一致
__global__ void visscal(
    int uvw_index, int lmnC_index,
    Complex* __restrict__ viss,    
    const float* __restrict__ u,   
    const float* __restrict__ v,
    const float* __restrict__ w,
    const float* __restrict__ l,
    const float* __restrict__ m,
    const float* __restrict__ n,
    const float* __restrict__ C,
    const Complex I1,             
    const Complex CPI,
    const Complex zero,
    const Complex two,
    const float dl,
    const float dm,
    const float dn)
{
    const int uvw_ = blockIdx.x * blockDim.x + threadIdx.x;
    if (uvw_ >= uvw_index) return;

    // 预先加载频繁使用的数据到寄存器
    const float u_val = u[uvw_] / dl;
    const float v_val = v[uvw_] / dm;
    const float w_val = w[uvw_] / dn;

    // 初始化累加器
    Complex acc = zero;
    // 主循环展开（编译器会进一步优化）
    #pragma unroll 8
    for (int lmnC_ = 0; lmnC_ < lmnC_index; ++lmnC_) {
        // 计算相位
        const float phase = u_val*l[lmnC_] + v_val*m[lmnC_] + w_val*(n[lmnC_]-1.0f);
        // 计算复指数
        const Complex exp_val = complexExp((zero - I1) * two * CPI * Complex(phase, 0.0f));
        // 累加结果
        acc += Complex(C[lmnC_], 0.0f) * exp_val;
    }
    
    // 计算最终的复指数因子
    const Complex final_exp = complexExp((zero - I1) * two * CPI * Complex(w_val, 0.0f));
    // 存储最终结果
    viss[uvw_] = acc * final_exp;
}


// 定义图像反演核函数  验证正确
__global__ void imagerecon(
    const int uvw_index,
    const int lmnC_index,
    Complex* __restrict__ F,                    
    const Complex* __restrict__ viss,           
    const float* __restrict__ u,
    const float* __restrict__ v,
    const float* __restrict__ w,
    const float* __restrict__ l,
    const float* __restrict__ m,
    const float* __restrict__ n,
    const float* __restrict__ C,
    const float* __restrict__ uvwFrequencyMap,
    const Complex I1,                    
    const Complex CPI,
    const Complex zero,
    const Complex two,
    const float dl,
    const float dm,
    const float dn)
{
    const int lmnC_ = blockIdx.x * blockDim.x + threadIdx.x;
    if (lmnC_ >= lmnC_index) return;

    // 预计算常量
    const Complex amount(uvw_index, 0.0f);      // 转换为常量
    // 预加载当前位置的 l, m, n 值到寄存器
    const float l_val = l[lmnC_] / dl;
    const float m_val = m[lmnC_] / dm;
    const float n_val = n[lmnC_] / dn;

    // 使用复数累加器
    Complex acc = zero;
    // 主循环
    #pragma unroll 8
    for (int uvw_ = 0; uvw_ < uvw_index; ++uvw_) {
        // 计算相位
        const float phase = u[uvw_]*l_val + v[uvw_]*m_val + w[uvw_]*n_val;
        // 计算复指数
        const Complex exp_val = complexExp(I1 * two * CPI * Complex(phase, 0.0f));
        // 累加结果
        acc += uvwFrequencyMap[uvw_] * viss[uvw_] * exp_val;
    }
    // 归一化并存储结果
    F[lmnC_] = acc / amount;
}


int vissGen(int id, int RES, int start_period) 
{   
    cout << "res: " << RES << endl;
    int days = 1;  // 一共有多少个周期  15月 * 30天 / 14天/周期
    cout << "periods: " << days << endl;
    Complex I1(0.0, 1.0);
    float dl = 2 * RES / (RES - 1);
    float dm = 2 * RES / (RES - 1);
    float dn = 2 * RES / (RES - 1);
    Complex zero(0.0, 0.0);
    Complex two(2.0, 0.0);
    Complex CPI(M_PI, 0.0);

    gettimeofday(&start, NULL);
    int nDevices;
    // 设置节点数量（gpu显卡数量）
    CHECK(hipGetDeviceCount(&nDevices));
    // 设置并行区中的线程数
    omp_set_num_threads(nDevices);
    cout << "devices: " << nDevices << endl;

    // 加载存储 l m n C nt的文件（对于不同的frequency不一样，只与frequency有关）
    string para, address_l, address_m, address_n, address_nt, address_NX, address_FF;
    ifstream lFile, mFile, nFile, ntFile, NXFile, FFFile;
    para = "l";
    address_l = address + para + sufix;
    lFile.open(address_l);
    cout << "address_l: " << address_l << endl;
    para = "m";
    address_m = address + para + sufix;
    mFile.open(address_m);
    cout << "address_m: " << address_m << endl;
    para = "n";
    address_n = address + para + sufix;
    nFile.open(address_n);
    cout << "address_n: " << address_n << endl;
    para = "NX";
    address_NX = address + para + sufix;
    NXFile.open(address_NX);
    cout << "address_NX: " << address_NX << endl;
    para = "FF";
    address_FF = address + para + sufix;
    FFFile.open(address_FF);
    cout << "address_FF: " << address_FF << endl;
    if (!lFile.is_open() || !mFile.is_open() || !nFile.is_open() || !NXFile.is_open() ||!FFFile.is_open()) {
        std::cerr << "无法打开一个或多个文件：" << std::endl;
        if (!lFile.is_open()) std::cerr << "无法打开文件: " << address_l << std::endl;
        if (!mFile.is_open()) std::cerr << "无法打开文件: " << address_m << std::endl;
        if (!nFile.is_open()) std::cerr << "无法打开文件: " << address_n << std::endl;
        if (!NXFile.is_open()) std::cerr << "无法打开文件: " << address_NX << std::endl;
        if (!FFFile.is_open()) std::cerr << "无法打开文件: " << address_FF << std::endl;
        return -1; 
    }
    int lmnC_index = 0;
    int NX_index = 0;
    lFile >> lmnC_index;  // 读取l的第一行的行数
    FFFile >> NX_index;  // 读取FF的第一行的行数
    cout << "lmnC index: " << lmnC_index << endl;
    cout << "NX index: " << NX_index << endl;

    std::vector<float> cl(lmnC_index), cm(lmnC_index), cn(lmnC_index);
    std::vector<float> cNX(NX_index), cFF(NX_index);
    for (int i = 0; i < lmnC_index && lFile.good() && mFile.good() && nFile.good() && ntFile.good(); ++i) {
        lFile >> cl[i];
        mFile >> cm[i];
        nFile >> cn[i];
    }
    for (int i = 0; i < NX_index && NXFile.good() && FFFile.good(); ++i) {
        NXFile >> cNX[i];
        FFFile >> cFF[i];
    }
    lFile.close();
    mFile.close();
    nFile.close();
    ntFile.close();
    NXFile.close();
    FFFile.close();

    // 导入uvw坐标的出现频次，txt文件的每一行每个坐标的频次
    auto uvwMapStart = std::chrono::high_resolution_clock::now();
    // 创建map存储
    std::unordered_map<std::string, float> cUVWFrequencyMap;
    string uvwmap_address = address + "uvwMap130.txt";
    std::ifstream uvwMapFile(uvwmap_address);
    if (uvwMapFile.is_open()) {
        // 读取第一行获取总行数
        string firstLine;
        std::getline(uvwMapFile, firstLine);
        int totalLines = std::stoi(firstLine);
        cout << "uvwMap totalLines: " << totalLines << endl;
        // 预分配内存
        cUVWFrequencyMap.reserve(totalLines);
        // 每一行的格式： -23 -288 -166 4
        string line;
        while (std::getline(uvwMapFile, line)) {
            std::istringstream iss(line);
            int u_point, v_point, w_point;
            int uvw_frequency;
            if (iss >> u_point >> v_point >> w_point >> uvw_frequency) {
                std::string key = std::to_string(u_point) + "_" + std::to_string(v_point) + "_" + std::to_string(w_point);
                cUVWFrequencyMap[key] = uvw_frequency;
            } else {
                cout << "Failed to parse line: " << line << endl; // 解析失败时的调试信息
            }
        }
        uvwMapFile.close();
    }
    // 打印测试确保是正确的
    int count = 0;
    int numElementsToPrint = 6; // 设定要打印的元素数量
    for (const auto& pair : cUVWFrequencyMap) {
        std::cout << pair.first << ": " << pair.second << std::endl;
        if (++count == numElementsToPrint) {
            break;
        }
    }
    cout << "Transfer uvw Frequency Success! " << endl;
    auto uvwMapFinish = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> uvwMapElapsed = uvwMapFinish - uvwMapStart;
    cout << "Transfer uvw Frequency Elapsed Time: " << uvwMapElapsed.count() << " s\n";

    // 读取C, 如果没有，则计算C
    std::vector<float> C_host(lmnC_index);
    ifstream cFile;
    string address_C = address + "C" + duration + sufix;
    cFile.open(address_C);
    if (cFile.is_open()) {
        cout << "C file is opened!" << endl;
        // 读取C文件
        for (int i = 0; i < lmnC_index && cFile.good(); ++i) {
            cFile >> C_host[i];
        }
        cFile.close();
    } else {
        cout << "C file is not exists, now compute C on GPU 0" << endl;
    
        // 使用一个GPU计算处理得到C
        auto computeCStart = std::chrono::high_resolution_clock::now();
        CHECK(hipSetDevice(0)); 
        thrust::device_vector<float> dNX = cNX;
        thrust::device_vector<float> dFF = cFF;
        thrust::device_vector<float> dC(lmnC_index);
        thrust::device_vector<float> dN = cn;
        // 调用CUDA核函数计算C
        int blockSize;
        int minGridSize; // 最小网格大小
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeC, 0, 0);
        int gridSize = (lmnC_index + blockSize - 1) / blockSize;;    // 线程块的数量
        cout << "Calculate C, blockSize: " << blockSize << endl;
        cout << "Calculate C, girdSize: " << gridSize << endl;
        computeC<<<gridSize, blockSize>>>(
            thrust::raw_pointer_cast(dNX.data()),
            thrust::raw_pointer_cast(dFF.data()),
            thrust::raw_pointer_cast(dC.data()),
            thrust::raw_pointer_cast(dN.data()),
            lmnC_index, NX_index
        );
        CHECK(hipDeviceSynchronize());
        std::cout << "C is computed in GPU 0!" << std::endl;

        // 将计算得到的C复制回主机
        thrust::copy(dC.begin(), dC.end(), C_host.begin());

        auto computeCFinish = std::chrono::high_resolution_clock::now();
        std::chrono::duration<float> computeCElapsed = computeCFinish - computeCStart;
        cout << "Compute C Elapsed Time: " << computeCElapsed.count() << " s\n";
        dNX.clear();
        dNX.shrink_to_fit();
        dFF.clear();
        dFF.shrink_to_fit();
        dC.clear();
        dC.shrink_to_fit();
        dN.clear();
        dN.shrink_to_fit();

        // 打开文件
        cout << "Save C to file " << address_C << " ..." << endl;
        cout << "save address_C: " << address_C << endl;
        std::ofstream file(address_C);
        if (file.is_open()) {
            // 按照指定格式写入文件
            for(const float& value : cNX)
            {
                file << value << std::endl;
            }
        }
        // 关闭文件
        file.close();
        std::cout << "save NX success!" << std::endl;
    }

    // 开启cpu线程并行
    // 一个线程处理1个GPU
    #pragma omp parallel
    {
        int tid = omp_get_thread_num();  // 从 0 开始编号的并行执行线程
        hipSetDevice(tid);
        std::cout << "Thread " << tid << " is running on device " << tid << std::endl;

        // 遍历所有开启的线程处理， 一个线程控制一个GPU 处理一个id*amount/total的块
        for (int p = tid+start_period; p < days; p += nDevices) 
        {
            cout << "for loop: " << p+1 << endl;

            // 将 l m n C NX 数据从cpu搬到GPU上        
            thrust::device_vector<float> l(cl.begin(), cl.end());
            thrust::device_vector<float> m(cm.begin(), cm.end());
            thrust::device_vector<float> n(cn.begin(), cn.end());
            thrust::device_vector<float> C(C_host.begin(), C_host.end());

            // 创建用来存储不同index中【u, v, w】
            std::vector<float> cu(uvw_presize), cv(uvw_presize), cw(uvw_presize);
            thrust::device_vector<float> u(uvw_presize), v(uvw_presize), w(uvw_presize);

            // 创建存储uvw坐标对应的频次
            std::vector<float> uvwMapVector(uvw_presize);
            thrust::device_vector<float> uvwFrequencyMap(uvw_presize);
        
            // 存储计算后的到的最终结果
            thrust::device_vector<Complex> F(lmnC_index);

            // 计时统计
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            // 记录开始事件
            hipEventRecord(start);

            // 记录uvw开始事件
            hipEvent_t uvwstart, uvwstop;
            hipEventCreate(&uvwstart);
            hipEventCreate(&uvwstop);
            hipEventRecord(uvwstart);

            // 创建一个临界区，保证只有一个线程进入，用于构建u v w
            int uvw_index;
            #pragma omp critical
            {
                string address_uvw = address + "uvw" + to_string(p+1) + duration + sufix;
                cout << "address_uvw: " << address_uvw << std::endl;
                
                ifstream uvwFile(address_uvw);
                // 同时用一个向量保存每一个uvw坐标点的frequency
                uvw_index = 0;
                float u_point, v_point, w_point;
                string key_point;
                if (uvwFile.is_open()) {
                    while (uvwFile >> u_point >> v_point >> w_point) {
                        // 直接构造 key_point
                        key_point = std::to_string(static_cast<int>(u_point)) + "_" + 
                                    std::to_string(static_cast<int>(v_point)) + "_" + 
                                    std::to_string(static_cast<int>(w_point));

                        // 简化查找操作
                        auto it = cUVWFrequencyMap.find(key_point);
                        if (it != cUVWFrequencyMap.end()) {
                            uvwMapVector[uvw_index] = 1 / (it->second);  // 存储频次的倒数
                        } else {
                            uvwMapVector[uvw_index] = 1; 
                        }
                        // cu, cv, cw 需要存储原始坐标
                        cu[uvw_index] = u_point;
                        cv[uvw_index] = v_point;
                        cw[uvw_index] = w_point;
                        uvw_index++;
                    }
                }               
                cout << "uvw_index: " << uvw_index << endl; 
                
                // 复制到GPU上
                thrust::copy(cu.begin(), cu.begin() + uvw_index, u.begin());
                thrust::copy(cv.begin(), cv.begin() + uvw_index, v.begin());
                thrust::copy(cw.begin(), cw.begin() + uvw_index, w.begin());
                thrust::copy(uvwMapVector.begin(), uvwMapVector.begin() + uvw_index, uvwFrequencyMap.begin());
            }

            // 记录uvw结束事件
            hipEventRecord(uvwstop);
            hipEventSynchronize(uvwstop);
            // 计算经过的时间
            float uvwMS = 0;
            hipEventElapsedTime(&uvwMS, uvwstart, uvwstop);
            printf("Period %d Load UWV Cost Time is: %f s\n", p+1, uvwMS/1000);
            // 销毁事件
            hipEventDestroy(uvwstart);
            hipEventDestroy(uvwstop);


            // 记录viss开始事件
            hipEvent_t vissstart, vissstop;
            hipEventCreate(&vissstart);
            hipEventCreate(&vissstop);
            hipEventRecord(vissstart);

            // 存储计算后的可见度
            thrust::device_vector<Complex> viss(uvw_index);
            int blockSize;
            int minGridSize; // 最小网格大小
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, visscal, 0, 0);
            int gridSize = floor(uvw_index + blockSize - 1) / blockSize;;  
            cout << "Viss Computing, blockSize: " << blockSize << endl;
            cout << "Viss Computing, girdSize: " << gridSize << endl;
            printf("Viss Computing... Here is gpu %d running process %d on node %d\n", omp_get_thread_num(), p+1, id);
            // 调用函数计算可见度

            visscal<<<gridSize, blockSize>>>(uvw_index, lmnC_index,
                    thrust::raw_pointer_cast(viss.data()),
                    thrust::raw_pointer_cast(u.data()),
                    thrust::raw_pointer_cast(v.data()),
                    thrust::raw_pointer_cast(w.data()),
                    thrust::raw_pointer_cast(l.data()),
                    thrust::raw_pointer_cast(m.data()),
                    thrust::raw_pointer_cast(n.data()),
                    thrust::raw_pointer_cast(C.data()),
                    I1, CPI, zero, two, dl, dm, dn);
            // 进行线程同步
            CHECK(hipDeviceSynchronize());
            cout << "period" << p+1 << " viss compute success!" << endl;

            // 记录viss结束事件
            hipEventRecord(vissstop);
            hipEventSynchronize(vissstop);
            // 计算经过的时间
            float vissMS = 0;
            hipEventElapsedTime(&vissMS, vissstart, vissstop);
            printf("Period %d Compute Viss Cost Time is: %f s\n", p+1, vissMS/1000);
            // 销毁事件
            hipEventDestroy(vissstart);
            hipEventDestroy(vissstop);


            // 记录imagerecon开始事件
            hipEvent_t imagereconstart, imagereconstop;
            hipEventCreate(&imagereconstart);
            hipEventCreate(&imagereconstop);
            hipEventRecord(imagereconstart);

            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, imagerecon, 0, 0);
            gridSize = floor(lmnC_index + blockSize - 1) / blockSize;
            cout << "Image Reconstruction, blockSize: " << blockSize << endl;
            cout << "Image Reconstruction, girdSize: " << gridSize << endl;
            printf("Image Reconstruction... Here is gpu %d running process %d on node %d\n",omp_get_thread_num(),p+1,id);
            // 调用image_recon函数计算图像反演
            imagerecon<<<gridSize,blockSize>>>(
                uvw_index, lmnC_index, 
                thrust::raw_pointer_cast(F.data()),
                thrust::raw_pointer_cast(viss.data()),
                thrust::raw_pointer_cast(u.data()),
                thrust::raw_pointer_cast(v.data()),
                thrust::raw_pointer_cast(w.data()),
                thrust::raw_pointer_cast(l.data()),
                thrust::raw_pointer_cast(m.data()),
                thrust::raw_pointer_cast(n.data()),
                thrust::raw_pointer_cast(C.data()),
                thrust::raw_pointer_cast(uvwFrequencyMap.data()),
                I1, CPI, zero, two, dl, dm, dn);
            // 进行线程同步
            CHECK(hipDeviceSynchronize());
            cout << "Period " << p+1 << "Image Reconstruction Success!" << endl;
            
            // 记录imagerecon结束事件
            hipEventRecord(imagereconstop);
            hipEventSynchronize(imagereconstop);
            // 计算经过的时间
            float imagereconMS = 0;
            hipEventElapsedTime(&imagereconMS, imagereconstart, imagereconstop);
            printf("Period %d Image Reconstruction Cost Time is: %f s\n", p+1, imagereconMS/1000);
            // 销毁事件
            hipEventDestroy(imagereconstart);
            hipEventDestroy(imagereconstop);


            // 记录saveimage开始事件
            hipEvent_t saveimagestart, saveimagestop;
            hipEventCreate(&saveimagestart);
            hipEventCreate(&saveimagestop);
            hipEventRecord(saveimagestart);
            // 创建一个临界区，用于保存图像反演结果
            #pragma omp critical
            {   
                // 在CPU上创建变量保存F结果
                thrust::host_vector<Complex> tempF = F;
                thrust::host_vector<Complex> extendF(NX_index);

                std::ofstream F_File;
                string address_F = "F_recon_1M/F" + to_string(p+1) + "_optim1.txt";
                cout << "address_F: " << address_F << endl;
                F_File.open(address_F);
                if (!F_File.is_open()) {
                    std::cerr << "Error opening file: " << address_F << endl;
                }
                for (int c = 0; c < NX_index; c++) {
                    int tmp = static_cast<int>(cNX[c]) - 1;  // matlab中下标从1开始
                    extendF[c] = tempF[tmp];
                    F_File << extendF[c].real() << std::endl;
                }
                F_File.close();
                std::cout << "Period " << p+1 << " save F success!" << std::endl;
            }

            // 记录saveimage结束事件
            hipEventRecord(saveimagestop);
            hipEventSynchronize(saveimagestop);
            // 计算经过的时间
            float saveimageMS = 0;
            hipEventElapsedTime(&saveimageMS, saveimagestart, saveimagestop);
            printf("Period %d Save Image Cost Time is: %f s\n", p+1, saveimageMS/1000);
            // 销毁事件
            hipEventDestroy(saveimagestart);
            hipEventDestroy(saveimagestop);

            // 记录结束事件
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            // 计算经过的时间
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("Period %d Elapsed time: %f s\n", p+1, milliseconds/1000);
            // 销毁事件
            hipEventDestroy(start);
            hipEventDestroy(stop);
        }
    }
    
    gettimeofday(&finish, NULL);
    total_time = ((finish.tv_sec - start.tv_sec) * 1000000 + (finish.tv_usec - start.tv_usec)) / 1000000.0;
    cout << "total time: " << total_time << "s" << endl;
    return 0;
}


int main()
{   
    int start_period = 0;  // 从哪个周期开始，一共是130个周期
    vissGen(0, 2094, start_period);
}

